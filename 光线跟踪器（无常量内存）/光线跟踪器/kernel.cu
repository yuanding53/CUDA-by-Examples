#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <cpu_bitmap.h>

#define INF 2e10f
#define rnd(x)(x*rand()/RAND_MAX)
#define SPHERES 20
#define DIM 1024

struct Sphere {
	float r, b, g;
	float radius;
	float x, y, z;
	__device__ float hit(float ox, float oy, float *n) {
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy*dy < radius*radius) {
			float dz = sqrtf(radius*radius - dx*dx - dy*dy);
			*n = dz / sqrtf(radius*radius);
			return dz + z;
		}
		return -INF;
	}
};

struct DataBlock {
	unsigned char   *dev_bitmap;
	Sphere          *s;
};

__global__ void kernel(Sphere *s,unsigned char *ptr) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y*blockDim.x*gridDim.x;
	float ox = (x - DIM / 2);
	float oy = (y - DIM / 2);

	float r = 0, g = 0, b = 0;
	float maxz = -INF;
	for (int i = 0; i < SPHERES; i++) {
		float n;
		float t = s[i].hit(ox, oy, &n);
		if (t > maxz) {
			float fscale = n;
			r = s[i].r*fscale;
			g = s[i].g*fscale;
			b = s[i].b*fscale;
		}
	}
	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}
	
int main(void) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	DataBlock data;
	CPUBitmap bitmap(DIM, DIM, &data);
	unsigned char *dev_bitmap;
	Sphere *s;

	hipMalloc((void**)&dev_bitmap, bitmap.image_size());
	hipMalloc((void**)&s, sizeof(Sphere)*SPHERES);

	Sphere *temp_s = (Sphere*)malloc(sizeof(Sphere)*SPHERES);
	for (int i = 0; i < SPHERES; i++) {
		temp_s[i].r = rnd(1.0f);
		temp_s[i].g = rnd(1.0f);
		temp_s[i].b = rnd(1.0f);
		temp_s[i].x = rnd(1000.0f) - 500;
		temp_s[i].y = rnd(1000.0f) - 500;
		temp_s[i].z = rnd(1000.0f) - 500;
		temp_s[i].radius = rnd(100.0f) + 20;
	}
	hipMemcpy(s, temp_s, sizeof(Sphere)*SPHERES, hipMemcpyHostToDevice);
	free(temp_s);

	dim3 grids(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel << <grids, threads >> > (s,dev_bitmap);
	hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);
	bitmap.display_and_exit();

	hipFree(dev_bitmap);
	hipFree(s);

};