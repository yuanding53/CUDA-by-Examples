#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include "cpu_bitmap.h"
#include <cpu_anim.h>

#define DIM 960

struct DataBlock {
	unsigned char *dev_bitmap;
	CPUAnimBitmap *bitmap;
};

void cleanup(DataBlock *d) {
	hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char *ptr, int ticks) {
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	int y = threadIdx.y + blockIdx.y*blockDim.y;
	int offset = x + y*blockDim.x*gridDim.x;

	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx*fx + fy*fy);
	unsigned char gray = (unsigned char)(128.0f + 127.0f*cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

	ptr[offset * 4 + 0] = gray;
	ptr[offset * 4 + 1] = gray;
	ptr[offset * 4 + 2] = gray;
	ptr[offset * 4 + 3] = 255;
}

void generate_frame(DataBlock *d, int ticks) {
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
	kernel << <blocks, threads >> > (d->dev_bitmap, ticks);
	hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost);
}

int main(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	hipMalloc((void**)&data.dev_bitmap, bitmap.image_size());
	bitmap.anim_and_exit((void(*)(void*, int))generate_frame, (void(*)(void*))cleanup);
}