#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>

#define imin(a,b)(a<b?a:b);

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while (tid < N) {
		temp += a[tid] * b[tid];
		tid += blockDim.x*gridDim.x;
	}

	cache[cacheIndex] = temp;
	
	__syncthreads();

	int i = blockDim.x / 2;
	while (i != 0) {
		if(cacheIndex<i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(void) {
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float));

	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
	
	dot << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_partial_c);

	hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

	c = 0;
	for (int i = 0; i < blocksPerGrid; i++) {
		c += partial_c[i];
	}

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
	printf("Does GPU value %.6g=%.6g ?\n", c, 2 * sum_squares((float)(N - 1)));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);

	free(a);
	free(b);
	free(partial_c);
}






